#include <stdio.h>
#include "hip/hip_runtime.h"
int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("%d", prop.major * 10 + prop.minor);
}
